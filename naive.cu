#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <string>
#include <sstream>
#include <fstream>
#include<iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unordered_map>
typedef unsigned long long ll;

__global__ void search_kernel(ll* frequency, const short* data, const ll len, const ll size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > size) return;
    int res = 0;
    for (int j = 0; j < size - len + 1; ++j) {
        int is_eq = 1;
        for (int k = 0; k < len && is_eq; ++k) {
            if (data[i + k] != data[j + k]) is_eq = 0;
        }
        res += is_eq;
    }
    frequency[i] = res;
}

int main(int argc, char* argv[]) {
    std::unordered_map<char, char> mapSymbToCode = { {'A', (char)0}, {'C', (char)1}, {'G', (char)2}, {'T', (char)3} };
    float exe_milliseconds = 0;
    float copy_milliseconds = 0;
    hipEvent_t start, exe_stop, copy_stop, start_copy;
    hipEventCreate(&start);
    hipEventCreate(&start_copy);
    hipEventCreate(&exe_stop);
    hipEventCreate(&copy_stop);

    ll len = std::atoi(argv[2]);
    std::ofstream fout("tmp.txt");
    FILE* data_file = fopen(argv[1], "rb");

    fseek(data_file, 0, SEEK_END);
    ll fsize = ftell(data_file);
    fseek(data_file, 0, SEEK_SET);
    char* data_h = (char*)malloc(fsize);


    fread(data_h, fsize, 1, data_file);

    short* enc_data_h = (short*)malloc(sizeof(short) * fsize);

    for (int i = 0; i < fsize; ++i) enc_data_h[i] = mapSymbToCode[data_h[i]];

    short* enc_data_d;
    hipMalloc((void**)&enc_data_d, sizeof(short) * fsize);
    hipMemcpy(enc_data_d, enc_data_h, fsize * sizeof(short), hipMemcpyHostToDevice);
    ll* frequency_dev;
    hipMalloc((void**)&frequency_dev, fsize * sizeof(ll));

    ll* frequency_host = (ll*)malloc(fsize * sizeof(ll));
    

    
    hipEventRecord(start, 0);
    hipEventRecord(start_copy, 0);
    search_kernel << <(fsize - len + 256) / 256, 256 >> > (frequency_dev, enc_data_d, len, fsize);

    hipDeviceSynchronize();

    hipEventRecord(exe_stop, 0);


    hipEventSynchronize(exe_stop);

    hipEventElapsedTime(&exe_milliseconds, start, exe_stop);

    hipMemcpy(frequency_host, frequency_dev, fsize * sizeof(ll), hipMemcpyDeviceToHost);


    hipEventRecord(copy_stop, 0);
    hipEventSynchronize(copy_stop);
    hipEventElapsedTime(&copy_milliseconds, start_copy, copy_stop);
    std::cout<<exe_milliseconds<<" "<<copy_milliseconds;
}