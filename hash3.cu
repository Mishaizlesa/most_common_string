#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <string>
#include <sstream>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unordered_map>
typedef unsigned long long ll;


__global__ void search_kernel(ll* frequency, const short* data ,const ll len, const ll size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > size) return;
    int shift[64];
    for (int j = 0; j < 64; ++j) shift[j] = len - 2;
    int res = 0;
    int sh1;
    ll hash = 0;
    for (int j = 2; j <= len - 1; ++j) {
        int ind = (data[i + j - 2]) * 16 + (data[i + j - 1]) * 4 + (data[i + j]);
        if (j == len - 1) sh1 = shift[ind];
        shift[ind] = len - 1 - j;
    }

    if (!sh1) sh1 = 1;
    int j = len - 1;
    for (;;) {
        int sh = 1;
        while (sh && j <= size - len) {
            int ind = (data[j - 2]) * 16 + (data[j - 1]) * 4 + (data[j]);
            sh = shift[ind];
            j += sh;
        }
        if (j <= size - len) {
            int is_eq = 1;
            for (int k = 0; k < len; ++k) {
                if (data[i + k] != data[j - len + 1 + k]) {
                    is_eq = 0;
                    break;
                }
            }
            res += is_eq;
            j += sh1;
        }
        else {
            break;
        }
    }
    frequency[i] = res;
}

int main(int argc, char* argv[]) {
    std::unordered_map<char, char> mapSymbToCode = { {'A', (char)0}, {'C', (char)1}, {'G', (char)2}, {'T', (char)3} };
    float exe_milliseconds = 0;
    float copy_milliseconds = 0;
    hipEvent_t start, exe_stop, copy_stop, start_copy;
    hipEventCreate(&start);
    hipEventCreate(&start_copy);
    hipEventCreate(&exe_stop);
    hipEventCreate(&copy_stop);

    ll len = std::atoi(argv[2]);
    std::ofstream fout("tmp.txt");
    FILE* data_file = fopen(argv[1], "rb");

    fseek(data_file, 0, SEEK_END);
    ll fsize = ftell(data_file);
    fseek(data_file, 0, SEEK_SET);
    char* data_h = (char*)malloc(fsize);


    fread(data_h, fsize, 1, data_file);

    short* enc_data_h = (short*)malloc(sizeof(short) * fsize);

    for (int i = 0; i < fsize; ++i) enc_data_h[i] = mapSymbToCode[data_h[i]];

    short* enc_data_d;
    hipMalloc((void**)&enc_data_d, sizeof(short) * fsize);
    hipMemcpy(enc_data_d, enc_data_h, fsize * sizeof(short), hipMemcpyHostToDevice);
    ll* frequency_dev;
    hipMalloc((void**)&frequency_dev, fsize*sizeof(ll) );

    ll* frequency_host = (ll*)malloc(fsize * sizeof(ll));
    hipEventRecord(start,0);
    hipEventRecord(start_copy, 0);
    search_kernel<< <(fsize - len + 256) / 256, 256 >> > (frequency_dev, enc_data_d, len, fsize);
    hipDeviceSynchronize();

    hipEventRecord(exe_stop,0);


    hipEventSynchronize(exe_stop);

    hipEventElapsedTime(&exe_milliseconds, start, exe_stop);
   
    hipMemcpy(frequency_host, frequency_dev, fsize * sizeof(ll), hipMemcpyDeviceToHost);


    hipEventRecord(copy_stop, 0);
    hipEventSynchronize(copy_stop);
    hipEventElapsedTime(&copy_milliseconds, start_copy, copy_stop);
    std::cout<<exe_milliseconds<<" "<<copy_milliseconds;
}